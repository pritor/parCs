#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>

#include <hip/hip_runtime.h>

#include "../Inc/FCLayer.cuh"
#include "../Inc/Errors.cuh"


// CUDA kernel to apply the sigmoid function to each element in the array
__global__
void _sigmoid(float* data, int size)
{
    size_t x = blockDim.x * blockIdx.x + threadIdx.x;

    if (x < size)
    {
        // Apply the sigmoid function to the current element
        data[x] = 1.0 / (1.0 + expf(-data[x]));
    }
}

// Host function to call the sigmoid kernel
__host__
void sigmoid(float* data, int size)
{
    // Set the number of threads per block
    size_t threads = 16;

    // Calculate the number of blocks needed based on the array size
    size_t blocks = std::ceil(1.0 * size / threads);

    // Set the block size and grid size for the kernel
    dim3 blockSize(threads);
    dim3 gridSize(blocks);

    // Launch the sigmoid kernel
    _sigmoid<<<gridSize, blockSize>>>(data, size);
}

LinearArguments::LinearArguments(const char* pathToWeights, const char* pathToBias, int inSize, int outSize) :
        _pathToWeights(pathToWeights), _pathToBias(pathToBias), _inSize(inSize), _outSize(outSize)  {}

const char* LinearArguments::getPathToWeights() { return this->_pathToWeights; }

const char* LinearArguments::getPathToBias() { return this->_pathToBias; }

int LinearArguments::getInputSize() { return this->_inSize; }

int LinearArguments::getOutputSize() { return this->_outSize; }


Linear::Linear(hipblasHandle_t handle, LinearArguments args) : cublasHandle(handle)
{
    this->sizeX = args.getOutputSize();
    this->sizeY = args.getInputSize();

    // Allocate memory
    float* tempBufferForWeights;
    float* tempBufferForBias;
    GET_CUDA_STATUS(hipHostMalloc(&tempBufferForWeights, sizeof(float) * this->sizeY * this->sizeX));
    GET_CUDA_STATUS(hipHostMalloc(&tempBufferForBias, sizeof(float)*this->sizeX));
    GET_CUDA_STATUS(hipMalloc(&this->weights, sizeof(float) * this->sizeY * this->sizeX));
    GET_CUDA_STATUS(hipMalloc(&this->bias, sizeof(float)  * this->sizeX));
    GET_CUDA_STATUS(hipMalloc(&this->output, sizeof(float) * this->sizeX));

    // Here we will write weights from 'pathToWeights' file
    FILE* f_in_weights = std::fopen(args.getPathToWeights(), "rb");
    if (!f_in_weights)
    {
        std::cout << "There's no such file: " << args.getPathToWeights() << std::endl;
        std::exit(-1);
    }

    FILE* f_in_bias = std::fopen(args.getPathToBias(), "rb");
    if (!f_in_bias)
    {
        std::cout << "There's no such file: " << args.getPathToBias() << std::endl;
        std::exit(-1);
    }

    std::fread(tempBufferForWeights, sizeof(float), this->sizeY * this->sizeX, f_in_weights);
    std::fread(tempBufferForBias, sizeof(float), this->sizeX, f_in_bias);

    GET_CUDA_STATUS(hipMemcpy(
            (void*)this->weights,
            (void*)tempBufferForWeights,
            sizeof(float) * this->sizeY * this->sizeX,
            hipMemcpyHostToDevice));
    GET_CUDA_STATUS(hipMemcpy(
            (void*)this->bias,
            (void*)tempBufferForBias,
            sizeof(float) * this->sizeX,
            hipMemcpyHostToDevice));

    // Delete temp buffer
    GET_CUDA_STATUS(hipHostFree(tempBufferForWeights));
    GET_CUDA_STATUS(hipHostFree(tempBufferForBias));
    std::fclose(f_in_weights);
    std::fclose(f_in_bias);
}

Linear::~Linear()
{
    if (this->output)   GET_CUDA_STATUS(hipFree(this->output));
    if (this->weights)  GET_CUDA_STATUS(hipFree(this->weights));
    if (this->bias)  GET_CUDA_STATUS(hipFree(this->bias));
}

void Linear::forward(float* input, float** output)
{
    const float alpha = 1.0, beta = 0.0;
    //matrix multiplying
    GET_CUBLAS_STATUS(hipblasSgemv(
            this->cublasHandle,
            HIPBLAS_OP_T,
            this->sizeY,
            this->sizeX,
            &alpha,
            this->weights,
            this->sizeY,
            input,
            1,
            &beta,
            this->output,
            1));
    //adding bias
    GET_CUBLAS_STATUS(hipblasSaxpy(this->cublasHandle, this->sizeX, &alpha, this->bias, 1, this->output, 1));
    *output = this->output;
}

int Linear::getInputSize()
{
    return this->sizeY;
}

int Linear::getOutputSize()
{
    return this->sizeX;
}